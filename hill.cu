#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ASCII_CHARS 128



void printrow(float *msg_v_list) {
    int i;
    for (i = 0; i < 16; i++) {
        printf("PRINTROW: %f\n", msg_v_list[i]);
    }

}

__global__ void mtxEncrypt(float *secretKey, float *msg, float *result, int matrix_dims) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float Pvalue = 0;
    int i;
    for (i = 0; i < 16; i++) {
       // printf("PRINTROW: %f\n", msg[i]);
    }
    for (int k = 0; k < matrix_dims; ++k) {
        float secretElement = secretKey[ty * matrix_dims + k];
        float msgElement = msg[k * matrix_dims + tx];
        
        printf("\nMultiplying %f by %f: \n", secretElement, msgElement);
        Pvalue  += secretElement * msgElement;
    }
    
    printf("PValue: %f\n", Pvalue);
    result[ty * matrix_dims + tx] = (int)Pvalue%ASCII_CHARS;
    printf("result: %f\n", result[ty * matrix_dims + tx]);
}

void secretKey(float **SKey, float **invSKey, int matrix_dims) {
    int k,z;

    float xSKey[4][4] = {{8, 6, 9, 5},
                {6, 9, 5, 10},
                {5, 8, 4, 9},
                {10, 6, 11, 4}};
    float xinvSKey[4][4] = {{-3, 20, -21, 1},
                    {2, -41, 44, 1},
                    {2, -6, 6, -1},
                    {-1, 28, -30, -1}};

    int i, j;
    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            SKey[0][i*matrix_dims + j] = xSKey[i][j];
            invSKey[0][i*matrix_dims + j] = xinvSKey[i][j];
            printf("%f :  %f \n", i*matrix_dims + j, SKey[0][i*matrix_dims + j]);
        }
    }
}

void pad_msg(char *msg, int matrix_dims) {
    int i;
    unsigned int msg_len = strlen(msg);
    int matrix_size = matrix_dims*matrix_dims;
    int extra_chars = matrix_size - msg_len%matrix_size;

    char *space = (char *)malloc(extra_chars);
    for (i = 0; i < extra_chars; i++) {
        space[i] = ' ';
    }
    strcat(msg, space);
}

void encode_msg(char *msg, float *msg_vector, int matrix_dims) {
    int matrix_size = matrix_dims * matrix_dims;
    int i = 0, j;

    while (msg[i] != '\0') {
        msg_vector[i] = (int)(msg[i]);
        i++;
    }
}

int main(int argc, char *argv[]) {

    char *msg;
    if (argc > 1) {

        if (fopen(argv[1], "r")) {
            printf("reading file\n\n");
            FILE *msg_file;
            msg_file = fopen(argv[1], "r");
            fseek(msg_file, 0, SEEK_END);
            // ftell() gives current position in the stream
            long msg_file_size = ftell(msg_file);
            // rewind to beginning of file now that we have size
            fseek(msg_file, 0, SEEK_SET);
            // allocate memory for msg var, read file stream into memory
            char *msg_file_text = (char *)malloc(msg_file_size + 1);
            fread(msg_file_text, msg_file_size, 1, msg_file);
            fclose(msg_file);
            // printf("%s", msg_file_text);
            msg = msg_file_text;
        } else {
            msg = (char *)malloc(strlen(argv[1]) + 15);
            strcpy(msg, argv[1]);
        }
    } else {
        const char* jack_msg = "All work and no play makes Jack a dull boy.";
        msg = (char *) malloc(strlen(jack_msg));
        strcpy(msg, jack_msg);
    }
    char *ascii_dict = (char *)malloc(128);

    //printf("%s", msg);
    int matrix_dims = 4;
    int matrix_size = matrix_dims*matrix_dims;
    // int **SKey;


    // malloc() for 4 (int*) pointers to 4 rows, also malloc'd (int)
    int z;
    float **SKey = (float **) malloc(matrix_dims * sizeof(float *));
    for (z = 0; z < matrix_dims; z++) {
        SKey[z] = (float *)malloc(matrix_dims * sizeof(float));
    }
    float **invSKey = (float **)malloc(matrix_dims * sizeof(float *));
    for (z = 0; z < matrix_dims; z++) {
        invSKey[z] = (float *)malloc(matrix_dims * sizeof(float));
    }

    int msg_size = strlen(msg);
    printf("STRLEN:  %d     MESSAGE SIZE: %d", strlen(msg), msg_size);
    // populate secret and inverse secret keys
    secretKey(SKey, invSKey, matrix_dims);

    int i, j;
    printf("%s", msg);
    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            printf("Skey: %f\n", SKey[0][i*4 + j]);
        }
    }
    // pad message if not divisible by matrix_dims^2
    if (msg_size % matrix_size != 0) {
        pad_msg(msg, matrix_dims);
        msg_size = strlen(msg);
    }
    printf("%s", msg);

    float *msg_vector = (float *)malloc(strlen(msg)*sizeof(float));
    // message encoded, results stored in msg_vector
    encode_msg(msg, msg_vector, matrix_dims);
    printf("ENCODED!\n");

    float msg_vector_list[msg_size/matrix_size][matrix_size];
    for (i = 0; i < msg_size/matrix_size; i++) {
        printf("\n\nCount: %d %d\n", i, msg_size%matrix_size);
        for (j = 0; j < matrix_size; j++) {
            msg_vector_list[i][j] = msg_vector[i*matrix_size + j];
            printf("%f\n", msg_vector_list[i][j], "\n");
        }
    }
    
    
    int nBytes = matrix_size*sizeof(float);
    printf("1\n");
    float **results = (float **)malloc(msg_size * sizeof(float *));
    printf("2\n");
    for (z = 0; z < msg_size/matrix_size; z++) {
        results[z] = (float *)malloc(matrix_size * sizeof(float));
        memset(results[z], 0, nBytes);
    }
   printf("3\n");

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    hipSetDevice(0);


    float *secretGpu, *msgGpu, *resultGpu;
    hipMalloc((void **)&secretGpu, nBytes);


    printf("msg_size/matrix_size: %d \n", msg_size/matrix_size);
    printf("\nmsg_vector_list[1]: %d\n", *msg_vector_list[1]);


    hipMalloc((void **)&resultGpu, nBytes);
    hipMalloc((void **)&msgGpu, nBytes);


    hipMemcpy(secretGpu, *SKey, nBytes, hipMemcpyHostToDevice);
    
    dim3 block(matrix_dims, matrix_dims);
    dim3 grid((matrix_dims+block.x-1)/block.x, (matrix_dims+block.y-1)/block.y);

    for (i = 0; i < msg_size/matrix_size; i++) {
        printf("\nlist: %f", *msg_vector_list[i]);
        // printrow(msg_vector_list[i]); this was a sanity check.  i am sane.
        
        // this seems to be sending the same row over and over.  why?
        hipMemcpy(msgGpu, msg_vector_list[i], nBytes, hipMemcpyHostToDevice);
        mtxEncrypt<<<grid, block>>>(secretGpu, msgGpu, resultGpu, matrix_dims);
        hipMemcpy(results[i], resultGpu, nBytes, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }

    //cudaFree(secretGpu);

    printf("\n\n5\n\n");
    for (i = 0; i < msg_size/matrix_size; i++) {
    //    printf("%d %f\n\n\n\n", i, results[i]);
        for (j = 0; j < matrix_size; j++) {
            printf("%c", char((int)results[i][j]));
        }
    }
    
    float **unEncrypted = (float **)malloc(3 * sizeof(float *));
    for (z = 0; z < msg_size/matrix_size; z++) {
        unEncrypted[z] = (float *)malloc(matrix_size * sizeof(float));
        memset(unEncrypted[z], 0, nBytes);
    }
     

    hipMemcpy(secretGpu, *invSKey, nBytes, hipMemcpyHostToDevice);
    for (i = 0; i < msg_size/matrix_size; i++) {
        printf("OOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOO\nOOOOOOOOOOOOO\nOOOOOOOOOO\n");
        hipMemcpy(msgGpu, results[i], nBytes, hipMemcpyHostToDevice);
        mtxEncrypt<<<grid, block>>>(secretGpu, msgGpu, resultGpu, matrix_dims);
        hipMemcpy(unEncrypted[i], resultGpu, nBytes, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
    printf("HELLO");
    for (i = 0; i < msg_size/matrix_size; i++) {
        for (j = 0; j < matrix_size; j++) {
            printf("%c", char((int)unEncrypted[i][j]));
        }
    }
    free(results);
    hipFree(secretGpu);
    hipFree(msgGpu);
    hipFree(resultGpu);
    hipDeviceReset();

    

}





